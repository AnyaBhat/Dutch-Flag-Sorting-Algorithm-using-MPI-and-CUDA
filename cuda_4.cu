#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_SIZE 6
#define MAX_SEGMENTS 256
#define INSERTION_SORT_THRESHOLD 32

__device__ inline void swap(int& a, int& b) {
    int temp = a;
    a = b;
    b = temp;
}

__device__ void insertion_sort(int* arr, int low, int high) {
    for (int i = low + 1; i <= high; i++) {
        int key = arr[i];
        int j = i - 1;
        while (j >= low && arr[j] > key) {
            arr[j + 1] = arr[j];
            j--;
        }
        arr[j + 1] = key;
    }
}

__device__ int partition(int* arr, int low, int high, int* lt, int* gt) {
    if (high - low < INSERTION_SORT_THRESHOLD) {
        insertion_sort(arr, low, high);
        *lt = low;
        *gt = high;
        return low;
    }

    // Choose pivot using median-of-three
    int mid = low + (high - low) / 2;
    if (arr[mid] < arr[low]) swap(arr[low], arr[mid]);
    if (arr[high] < arr[low]) swap(arr[low], arr[high]);
    if (arr[mid] < arr[high]) swap(arr[mid], arr[high]);
    
    int pivot = arr[high];
    int i = low;
    *lt = low;
    *gt = high;
    
    while (i <= *gt) {
        if (arr[i] < pivot) {
            swap(arr[*lt], arr[i]);
            (*lt)++;
            i++;
        } else if (arr[i] > pivot) {
            swap(arr[i], arr[*gt]);
            (*gt)--;
        } else {
            i++;
        }
    }
    
    return *lt;
}

__device__ void three_way_quicksort_kernel(int* arr, int low, int high) {
    while (low < high) {
        if (high - low < INSERTION_SORT_THRESHOLD) {
            insertion_sort(arr, low, high);
            return;
        }

        int lt, gt;
        partition(arr, low, high, &lt, &gt);
        
        // Tail recursion optimization
        if (lt - low < high - gt) {
            three_way_quicksort_kernel(arr, low, lt - 1);
            low = gt + 1;
        } else {
            three_way_quicksort_kernel(arr, gt + 1, high);
            high = lt - 1;
        }
    }
}

__global__ void parallel_three_way_quicksort(int* arr, int n, int segment_size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int segment_start = tid * segment_size;
    
    if (segment_start < n) {
        int segment_end = min(segment_start + segment_size - 1, n - 1);
        three_way_quicksort_kernel(arr, segment_start, segment_end);
    }
}

__device__ void merge(int* arr, int* temp, int left, int mid, int right) {
    int i = left, j = mid + 1, k = left;
    
    while (i <= mid && j <= right) {
        if (arr[i] <= arr[j]) {
            temp[k++] = arr[i++];
        } else {
            temp[k++] = arr[j++];
        }
    }
    
    while (i <= mid) temp[k++] = arr[i++];
    while (j <= right) temp[k++] = arr[j++];
    
    for (i = left; i <= right; i++) {
        arr[i] = temp[i];
    }
}

__global__ void parallel_merge(int* arr, int* temp, int n, int segment_size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int left = tid * (segment_size * 2);
    
    if (left < n) {
        int mid = min(left + segment_size - 1, n - 1);
        int right = min(left + (segment_size * 2) - 1, n - 1);
        if (mid < right) {
            merge(arr, temp, left, mid, right);
        }
    }
}

void check_cuda_error(hipError_t error, const char* function_name) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in %s: %s\n", function_name, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

int main() {
    int n;
    int* h_arr;
    int* d_arr;
    int* d_temp;
    hipError_t cuda_status;

    // Read input from file
    FILE* input_file = fopen("input.txt", "r");
    if (!input_file) {
        fprintf(stderr, "Error opening input file\n");
        return 1;
    }

    fscanf(input_file, "%d", &n);
    h_arr = (int*)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++) {
        fscanf(input_file, "%d", &h_arr[i]);
    }
    fclose(input_file);

    printf("Read %d elements from input file\n", n);

    // Allocate device memory with aligned access
    cuda_status = hipMalloc((void**)&d_arr, n * sizeof(int));
    check_cuda_error(cuda_status, "hipMalloc d_arr");
    cuda_status = hipMalloc((void**)&d_temp, n * sizeof(int));
    check_cuda_error(cuda_status, "hipMalloc d_temp");

    // Copy input data to device
    cuda_status = hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "hipMemcpy H2D");

    // Create CUDA events
    hipEvent_t start1, stop1, start2, stop2;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    float milliseconds1, milliseconds= 0;
    float mtime=0;
    // Calculate optimal segment size and grid dimensions
    int segment_size = (n + MAX_SEGMENTS - 1) / MAX_SEGMENTS;
    segment_size = max(segment_size, INSERTION_SORT_THRESHOLD);
    
    int num_segments = (n + segment_size - 1) / segment_size;
    int num_blocks = (num_segments + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Launch sorting kernel
    hipEventRecord(start1, 0);
    parallel_three_way_quicksort<<<num_blocks, BLOCK_SIZE>>>(d_arr, n, segment_size);
    hipDeviceSynchronize();
    hipEventRecord(stop1, 0);
	
    // Merge sorted segments
    for (int curr_size = segment_size; curr_size < n; curr_size *= 2) {
        int merge_blocks = (n + curr_size * 2 - 1) / (curr_size * 2);
        merge_blocks = (merge_blocks + BLOCK_SIZE - 1) / BLOCK_SIZE;
        hipEventRecord(start2, 0);
        parallel_merge<<<merge_blocks, BLOCK_SIZE>>>(d_arr, d_temp, n, curr_size);
        hipEventRecord(stop2, 0);
        hipEventElapsedTime(&milliseconds1, start2, stop2);
        hipDeviceSynchronize();
        mtime+=milliseconds1;
    }

    // Record time and check for errors
    
    
    
    hipEventElapsedTime(&milliseconds, start1, stop1);
	
    cuda_status = hipGetLastError();
    check_cuda_error(cuda_status, "kernel execution");

    // Copy result back to host
    cuda_status = hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    check_cuda_error(cuda_status, "hipMemcpy D2H");

    // Verify and write output
    bool is_sorted = true;
    for (int i = 1; i < n && is_sorted; i++) {
        if (h_arr[i] < h_arr[i-1]) is_sorted = false;
    }
    printf("Array is %s\n", is_sorted ? "sorted" : "not sorted");

    FILE* output_file = fopen("output.txt", "w");
    if (output_file) {
        fprintf(output_file, "%d\n", n);
        for (int i = 0; i < n; i++) {
            fprintf(output_file, "%d ", h_arr[i]);
        }
        fclose(output_file);
        printf("Sorted %d elements and wrote to output file\n", n);
        printf("Sorting time: %f milliseconds\n", milliseconds+mtime);
    }

    // Cleanup
    hipFree(d_arr);
    hipFree(d_temp);
    free(h_arr);
    hipEventDestroy(start1);
    hipEventDestroy(stop1);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    return 0;
}
